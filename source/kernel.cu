
#include <hip/hip_runtime.h>
#include <iterator>

#include <cuda_pipeline_primitives.h>

#define BLOCK_X 16
#define BLOCK_Y 8

hipGraphExec_t get_graphexec(
    float * d_dst, float * d_src, float * h_buffer,
    int width, int height, int stride,
    float sigma_spatial, float sigma_color, int radius,
    bool use_shared_memory);

template <bool use_shared_memory, bool has_ref>
__global__
__launch_bounds__(BLOCK_X * BLOCK_Y)
static void bilateral(
    float * __restrict__ dst, const float * __restrict__ src,
    int width, int height, int stride,
    float sigma_spatial_scaled, float sigma_color_scaled, int radius) {

    const int x = threadIdx.x + blockIdx.x * BLOCK_X;
    const int y = threadIdx.y + blockIdx.y * BLOCK_Y;

    float num {};
    float den {};

    if constexpr (use_shared_memory) {
        extern __shared__ float buffer[
            /* (1 + has_ref) * (2 * radius + BLOCK_Y) * (2 * radius + BLOCK_X) */];

        for (int cy = threadIdx.y; cy < 2 * radius + BLOCK_Y; cy += BLOCK_Y) {
            int sy = min(max(cy - static_cast<int>(threadIdx.y) - radius + y, 0), height - 1);
            for (int cx = threadIdx.x; cx < 2 * radius + BLOCK_X; cx += BLOCK_X) {
                int sx = min(max(cx - static_cast<int>(threadIdx.x) - radius + x, 0), width - 1);
                __pipeline_memcpy_async(
                    &buffer[cy * (2 * radius + BLOCK_X) + cx],
                    &src[sy * stride + sx],
                    4
                );
            }
        }

        if constexpr (has_ref) {
            for (int cy = threadIdx.y; cy < 2 * radius + BLOCK_Y; cy += BLOCK_Y) {
                int sy = min(max(cy - static_cast<int>(threadIdx.y) - radius + y, 0), height - 1);
                for (int cx = threadIdx.x; cx < 2 * radius + BLOCK_X; cx += BLOCK_X) {
                    int sx = min(max(cx - static_cast<int>(threadIdx.x) - radius + x, 0), width - 1);
                    __pipeline_memcpy_async(
                        &buffer[(2 * radius + BLOCK_Y + cy) * (2 * radius + BLOCK_X) + cx],
                        &src[(height + sy) * stride + sx],
                        4
                    );
                }
            }
        }

        __pipeline_commit();
        __pipeline_wait_prior(0);
        __syncthreads();

        if (x >= width || y >= height)
            return;

        const float center = buffer[
            (has_ref * (2 * radius + BLOCK_Y) + radius + threadIdx.y) * (2 * radius + BLOCK_X) +
            radius + threadIdx.x
        ]; // src[(has_ref * height + y) * stride + x];

        for (int cy = -radius; cy <= radius; ++cy) {
            int sy = cy + radius + threadIdx.y;

            for (int cx = -radius; cx <= radius; ++cx) {
                int sx = cx + radius + threadIdx.x;

                float value = buffer[(has_ref * (2 * radius + BLOCK_Y) + sy) * (2 * radius + BLOCK_X) + sx];

                float space = cy * cy + cx * cx;
                float range = (value - center) * (value - center);

                float weight = exp2f(space * sigma_spatial_scaled + range * sigma_color_scaled);

                if constexpr (has_ref) {
                    value = buffer[sy * (2 * radius + BLOCK_X) + sx];
                }

                num += weight * value;
                den += weight;
            }
        }
    } else {
        if (x >= width || y >= height)
            return;

        const float center = src[(has_ref * height + y) * stride + x];

        for (int cy = max(y - radius, 0); cy <= min(y + radius, height - 1); ++cy) {
            for (int cx = max(x - radius, 0); cx <= min(x + radius, width - 1); ++cx) {
                float value = src[(has_ref * height + cy) * stride + cx];

                float space = (y - cy) * (y - cy) + (x - cx) * (x - cx);
                float range = (value - center) * (value - center);

                float weight = exp2f(space * sigma_spatial_scaled + range * sigma_color_scaled);

                if constexpr (has_ref) {
                    value = src[cy * stride + cx];
                }

                num += weight * value;
                den += weight;
            }
        }
    }

    dst[y * stride + x] = num / den;
}

hipGraphExec_t get_graphexec(
    float * d_dst, float * d_src, float * h_buffer,
    int width, int height, int stride,
    float sigma_spatial_scaled, float sigma_color_scaled, int radius,
    bool use_shared_memory, bool has_ref
) {

    size_t pitch { stride * sizeof(float) };

    hipGraph_t graph;
    hipGraphCreate(&graph, 0);

    hipGraphNode_t n_HtoD;
    {
        hipMemcpy3DParms copy_params {};
        copy_params.srcPtr = make_hipPitchedPtr(
            h_buffer, pitch, width, height * (1 + has_ref));
        copy_params.dstPtr = make_hipPitchedPtr(
            d_src, pitch, width, height * (1 + has_ref));
        copy_params.extent = make_hipExtent(
            width * sizeof(float), height * (1 + has_ref), 1);
        copy_params.kind = hipMemcpyHostToDevice;

        hipGraphAddMemcpyNode(&n_HtoD, graph, nullptr, 0, &copy_params);
    }

    hipGraphNode_t n_kernel;
    {
        hipGraphNode_t dependencies[] { n_HtoD };

        void * kernelArgs[] {
            &d_dst, &d_src,
            &width, &height, &stride,
            &sigma_spatial_scaled, &sigma_color_scaled, &radius
        };

        hipKernelNodeParams kernel_params {};

        auto sharedMemBytes = static_cast<unsigned int>(
            (1 + has_ref) * (2 * radius + BLOCK_Y) * (2 * radius + BLOCK_X) * sizeof(float));
        bool useSharedMem = use_shared_memory && sharedMemBytes < 48 * 1024;

        kernel_params.func = (
            useSharedMem ?
            (has_ref ?
                reinterpret_cast<void *>(bilateral<true, true>) :
                reinterpret_cast<void *>(bilateral<true, false>)
            ) :
            (has_ref ?
                reinterpret_cast<void *>(bilateral<false, true>) :
                reinterpret_cast<void *>(bilateral<false, false>)
            )
        );
        kernel_params.blockDim = dim3(BLOCK_X, BLOCK_Y);
        kernel_params.gridDim = dim3(
            (width - 1) / BLOCK_X + 1,
            (height - 1) / BLOCK_Y + 1
        );
        kernel_params.sharedMemBytes = useSharedMem ? sharedMemBytes : 0;
        kernel_params.kernelParams = kernelArgs;

        hipGraphAddKernelNode(
            &n_kernel, graph,
            dependencies, std::size(dependencies),
            &kernel_params);
    }

    hipGraphNode_t n_DtoH;
    {
        hipGraphNode_t dependencies[] { n_kernel };

        hipMemcpy3DParms copy_params {};
        copy_params.srcPtr = make_hipPitchedPtr(
            d_dst, pitch, width, height);
        copy_params.dstPtr = make_hipPitchedPtr(
            h_buffer, pitch, width, height);
        copy_params.extent = make_hipExtent(
            width * sizeof(float), height, 1);
        copy_params.kind = hipMemcpyDeviceToHost;

        hipGraphAddMemcpyNode(
            &n_DtoH, graph,
            dependencies, std::size(dependencies),
            &copy_params);
    }

    hipGraphExec_t graphexecp;
    hipGraphInstantiate(&graphexecp, graph, nullptr, nullptr, 0);

    hipGraphDestroy(graph);

    return graphexecp;
}

